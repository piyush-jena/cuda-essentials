
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include <fstream>

struct timespec start, hostDataAlloc, cudaDataAlloc, cudaCalc, cudaCopyToHost, finish;
#define FACTOR 1e6

using namespace std;

__global__ void vectorAddition(int* A, int* B, int* R, int N, int NBlocks)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = blockDim.x*NBlocks;

    while (threadId < N)
    {
        R[threadId] = A[threadId] + B[threadId];
        threadId += stride;
    }
}

void verify_result(int* A, int* B, int* R, int N) 
{
    for (int i = 0; i < N; i++) 
    {
        assert(R[i] == A[i] + B[i]);
    }
}

int main(int argc, char* argv[])
{
    clock_gettime(CLOCK_MONOTONIC,&start);

    int *A, *B, *R;
    int *d_A, *d_B, *d_R;
    double time_usec = 0;

    int K = atoi(argv[1]);
    int N = K*FACTOR;

    int NUM_THREADS = atoi(argv[2]);
    int NUM_BLOCKS = 0;

    if (argc == 4)
        NUM_BLOCKS = atoi(argv[3]);
    else
        NUM_BLOCKS = (N + NUM_THREADS - 1)/NUM_THREADS;    
    
    size_t memorybytes = sizeof(int)*N;
	
	A = (int*) malloc(memorybytes);
	B = (int*) malloc(memorybytes);
	R = (int*) malloc(memorybytes);
	
	for(int i = 0; i < N; i++)
	{
		A[i] = rand() % 100;
		B[i] = rand() % 100;
	}

    clock_gettime(CLOCK_MONOTONIC,&hostDataAlloc);

    hipMalloc((void**)&d_A, memorybytes);
    hipMalloc((void**)&d_B, memorybytes);
    hipMalloc((void**)&d_R, memorybytes);

    hipMemcpy(d_A, A, memorybytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, memorybytes, hipMemcpyHostToDevice);
    hipMemcpy(d_R, R, memorybytes, hipMemcpyHostToDevice);

    clock_gettime(CLOCK_MONOTONIC,&cudaDataAlloc);

    vectorAddition<<<NUM_BLOCKS, NUM_THREADS>>>(d_A, d_B, d_R, N, NUM_BLOCKS);
    hipDeviceSynchronize();

    clock_gettime(CLOCK_MONOTONIC,&cudaCalc);

    hipMemcpy(R, d_R, memorybytes, hipMemcpyDeviceToHost);

    clock_gettime(CLOCK_MONOTONIC,&cudaCopyToHost);

    
    verify_result(A, B, R, N);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_R);

    free(A);
    free(B);
    free(R);

    clock_gettime(CLOCK_MONOTONIC,&finish);

    printf("q2 N=%d threads=%d blocks=%d \n",N,NUM_THREADS,NUM_BLOCKS);

    time_usec =(((double)finish.tv_sec *1000000 + (double)finish.tv_nsec/1000) - ((double)start.tv_sec *1000000 + (double)start.tv_nsec/1000));
	printf("Total execution time: %.03lf\n", time_usec/1000);

	time_usec =(((double)hostDataAlloc.tv_sec *1000000 + (double)hostDataAlloc.tv_nsec/1000) - ((double)start.tv_sec *1000000 + (double)start.tv_nsec/1000));
	printf("Total host Data Allocation Time: %.03lf\n", time_usec/1000);
	
	time_usec =(((double)cudaDataAlloc.tv_sec *1000000 + (double)cudaDataAlloc.tv_nsec/1000) - ((double)hostDataAlloc.tv_sec *1000000 + (double)hostDataAlloc.tv_nsec/1000));
	printf("Total Time to copy data to Device: %.03lf\n", time_usec/1000);

	time_usec =(((double)cudaCalc.tv_sec *1000000 + (double)cudaCalc.tv_nsec/1000) - ((double)cudaDataAlloc.tv_sec *1000000 + (double)cudaDataAlloc.tv_nsec/1000));
	printf("Total Time to perform calculations: %.03lf\n", time_usec/1000);

    time_usec =(((double)cudaCopyToHost.tv_sec *1000000 + (double)cudaCopyToHost.tv_nsec/1000) - ((double)cudaCalc.tv_sec *1000000 + (double)cudaCalc.tv_nsec/1000));
	printf("Total Time to copy data to host: %.03lf\n", time_usec/1000);

    return 0;
}
